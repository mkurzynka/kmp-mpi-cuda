
#include <hip/hip_runtime.h>
#include <algorithm> 
#include <iostream>
#include <cstring>
#include <fstream>
#include <sstream>
#include <string>
#include "time.h"

using namespace std;

void mpTable(char* pattern, int *P) {
    int m = strlen(pattern);
    int i, j;
 
    i = 0;
    j = P[0] = -1;

    while (i < m) {
       while (j > -1 && pattern[i] != pattern[j])
          j = P[j];
       P[++i] = ++j;
    }
}

void kmpTable(char* pattern, int *P) {
    int m = strlen(pattern);
    int k;
    
    P[0] = -1;

    for (int i = 1; i < m; i++) {
        k = P[i - 1];
        while (k >= 0) {
            if (pattern[k] == pattern[i - 1])
                break;
            else
                k = P[k];
        }
        P[i] = k + 1;
    }
}

//check whether target string contains pattern 
__global__ void kmpAlgorithm(char *text, char *pattern, int *P,int *pat_positions, int pattern_length, int text_length) {
    // get current cuda thread id
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    // assign "pointer" to beginning and end of this thread data chunk
    int i = pattern_length * index;
    int j = pattern_length * (index + 2) - 1;

    // if this is end of text then return
    if(i > text_length)
        return;

    // snap end pointer to end of text if it falls beyond
    if(j > text_length)
        j = text_length;

    int k = 0;        
    // int counter = 0; //xd
    // do kmp algorithm for chunk of text
    while (i < j) {
        if (k == -1) {
            i++;
            k = 0;
        } else if (text[i] == pattern[k]) {
            i++;
            k++;
            if (k == pattern_length) {
                pat_positions[i - pattern_length] = i - pattern_length;
                // pat_positions[counter] = i - pattern_length;
                i = i - k + 1;
                // counter++; //xd
            }
        } else
            k = P[k];
    }
    return;
}
 
int main(int argc, char* argv[]) {   
    
    // define nuber of cuda threads 
    int n_cuda_threads = 1024;

    // define variables
    bool is_kmp = 1;

    char *text_data;
    char *pattern;
    int *P;
    int *pat_positions;

    char *d_text_data;
    char *d_pattern;
    int *d_P;
    int *d_pat_positions;

    clock_t app_elapsed_time, t_start, t_end;

    
    // read text data file
    app_elapsed_time = clock();

    t_start = clock();
    std::ifstream file(argv[1]);

    if (file.fail()) {
        std::cout << argv[1] << " file does not exist, exiting" << std::endl;
        exit(1);
    }

    std::stringstream buffer;
    buffer << file.rdbuf();
    std::string string_buffer = buffer.str();
    text_data = (char*)string_buffer.c_str();

    t_end = clock();
    printf("Reading data elapsed time: %f s\n", ((double) t_end - t_start) / CLOCKS_PER_SEC);

    // initialize arrays for kmp algorithm
    pattern = argv[2];
    is_kmp = argv[3];

    int text_length = strlen(text_data);
    int pattern_length = strlen(pattern);

    P = new int[text_length];
    pat_positions = new int[text_length];

    std::fill_n(pat_positions, text_length, -1);   

    t_start = clock();

    // precompute mp or kmp table
    if(is_kmp)
        kmpTable(pattern, P);
    else
        mpTable(pattern, P);
 
    t_end = clock();
    printf("Table construction: %f s\n", ((double) t_end - t_start) / CLOCKS_PER_SEC);

    // transfer data from RAM to VRAM
    t_start = clock();

    hipMalloc((void **)&d_text_data, text_length*sizeof(char));
    hipMalloc((void **)&d_pattern, pattern_length*sizeof(char));
    hipMalloc((void **)&d_P, text_length*sizeof(int));
    hipMalloc((void **)&d_pat_positions, text_length*sizeof(int));

    hipMemcpy(d_text_data, text_data, text_length*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_pattern, pattern, pattern_length*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_P, P, text_length*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_pat_positions, pat_positions, text_length*sizeof(int), hipMemcpyHostToDevice);

    t_end = clock();
    printf("Data transfer to cuda: %f s\n", ((double) t_end - t_start) / CLOCKS_PER_SEC);

    float elapsed_time_gpu = 0;
    hipEvent_t start_time, stop_time;

    // Strat kmp algorithm on GPU
    hipEventCreate(&start_time);
    hipEventCreate(&stop_time);

    hipEventRecord(start_time, 0); 

    // kmpAlgorithm<<<(text_length/pattern_length + n_cuda_threads)/n_cuda_threads, n_cuda_threads>>>(d_text_data, d_pattern, d_P, d_pat_positions, pattern_length, text_length);
    kmpAlgorithm<<<text_length/pattern_length/n_cuda_threads + 1, n_cuda_threads>>>(d_text_data, d_pattern, d_P, d_pat_positions, pattern_length, text_length);

    hipEventRecord(stop_time, 0); 
 
    hipEventSynchronize(start_time);    
    hipEventSynchronize(stop_time);    
    hipEventElapsedTime(&elapsed_time_gpu, start_time, stop_time);  


    printf("KMP algorithm finished, elapsed time on gpu: %f s\n", elapsed_time_gpu/1000);  

    // Transfer data from VRAM to RAM
    t_start = clock();

    hipMemcpy(pat_positions, d_pat_positions, text_length*sizeof(int), hipMemcpyDeviceToHost);

    t_end = clock();
    printf("Data transfer from cuda: %f s\n", ((double) t_end - t_start) / CLOCKS_PER_SEC);

    // Count all occurrences
    t_start = clock();

    // Post-process results
    int occurrences = 0;
    for(int i = 0; i < text_length; i++) { 
        if(pat_positions[i] != -1) {
            occurrences++;
        } 
    }

    t_end = clock();
    printf("Post-processing: %f s\n", ((double) t_end - t_start) / CLOCKS_PER_SEC);

    app_elapsed_time = clock() - app_elapsed_time;

    printf("Number of occurences: %d, elapsed time: %f s\n", occurrences, ((double) app_elapsed_time) / CLOCKS_PER_SEC);

    // free RAM and VRAM variables
    hipFree(d_text_data); 
    hipFree(d_pattern);
    hipFree(d_P);
    hipFree(pat_positions);
    delete []P;
    delete []pat_positions;

    return 0;
}
